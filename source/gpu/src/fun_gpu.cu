#include "hip/hip_runtime.h"
#include "cst_gpu.cu"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>


/***********************************************************
 * Vars
 ***********************************************************/
texture<unsigned short int, 1, hipReadModeElementType> tex_phantom;
texture<float, 1, hipReadModeElementType> tex_act_val;
texture<unsigned int, 1, hipReadModeElementType> tex_act_ind;

__constant__ const float pi = 3.14159265358979323846;
__constant__ const float twopi = 2*pi;

// Stack of gamma particles, format data is defined as SoA
struct StackGamma{
	float* E;
	float* dx;
	float* dy;
	float* dz;
	float* px;
	float* py;
	float* pz;
	float* t;
	unsigned int* seed;
	unsigned char* interaction;
	unsigned char* live;
	unsigned char* endsimu;
	unsigned char* ct_cpt;
	unsigned char* ct_pe;
	unsigned char* ct_ray;
	unsigned int size;
	unsigned long* table_x_brent;
}; //


/***********************************************************
 * Utils Device
 ***********************************************************/

// function from CLHEP
__device__ float3 deflect_particle(float3 p, float3 dir) {
	float u1 = p.x;
	float u2 = p.y;
	float u3 = p.z;
	float up = u1*u1 + u2*u2;

	if (up>0) {
		up = sqrt(up);
		float px = dir.x,  py = dir.y,  pz = dir.z;
		dir.x = __fdividef(u1*u3*px - u2*py, up) + u1*pz;
		dir.y = __fdividef(u2*u3*px + u1*py, up) + u2*pz;
		dir.z =              -up*px +              u3*pz;
    }
	else if (u3 < 0.) { dir.x = -dir.x; dir.z = -dir.z; }      // phi=0  theta=pi

	return make_float3(dir.x, dir.y, dir.z);
}

/***********************************************************
 * PRNG Brent xor256
 ***********************************************************/

// Brent PRNG integer version
__device__ unsigned long weyl;
__device__ unsigned long brent_int(unsigned int index, unsigned long *device_x_brent, unsigned long seed)

{
	
#define UINT64 (sizeof(unsigned long)>>3)
#define UINT32 (1 - UINT64) 
#define wlen (64*UINT64 +  32*UINT32)
#define r    (4*UINT64 + 8*UINT32)
#define s    (3*UINT64 +  3*UINT32)
#define a    (37*UINT64 +  18*UINT32)
#define b    (27*UINT64 +  13*UINT32)
#define c    (29*UINT64 +  14*UINT32)
#define d    (33*UINT64 +  15*UINT32)
#define ws   (27*UINT64 +  16*UINT32) 

	int z, z_w, z_i_brent;	
	if (r==4){
		z=6; z_w=4; z_i_brent=5;}
	else{
		z=10; z_w=8; z_i_brent=9;}
	
	unsigned long w = device_x_brent[z*index + z_w];
	unsigned long i_brent = device_x_brent[z*index + z_i_brent];
	unsigned long zero = 0;
	unsigned long t, v;
	int k;
	
	if (seed != zero) { // Initialisation necessary
		// weyl = odd approximation to 2**wlen*(3-sqrt(5))/2.
		if (UINT32) 
			weyl = 0x61c88647;
		else 
			weyl = ((((unsigned long)0x61c88646)<<16)<<16) + (unsigned long)0x80b583eb;
		
		v = (seed!=zero)? seed:~seed;  // v must be nonzero
		
		for (k = wlen; k > 0; k--) {   // Avoid correlations for close seeds
			v ^= v<<10; v ^= v>>15;    // Recurrence has period 2**wlen-1
			v ^= v<<4;  v ^= v>>13;    // for wlen = 32 or 64
		}
		for (w = v, k = 0; k < r; k++) { // Initialise circular array
			v ^= v<<10; v ^= v>>15; 
			v ^= v<<4;  v ^= v>>13;
			device_x_brent[k + z*index] = v + (w+=weyl);              
		}
		for (i_brent = r-1, k = 4*r; k > 0; k--) { // Discard first 4*r results
			t = device_x_brent[(i_brent = (i_brent+1)&(r-1)) + z*index];   t ^= t<<a;  t ^= t>>b;			
			v = device_x_brent[((i_brent+(r-s))&(r-1)) + z*index];	v ^= v<<c;  v ^= v>>d;       
			device_x_brent[i_brent + z*index] = t^v;  
		}
    }
    
	// Apart from initialisation (above), this is the generator
	t = device_x_brent[(i_brent = (i_brent+1)&(r-1)) + z*index]; // Assumes that r is a power of two
	v = device_x_brent[((i_brent+(r-s))&(r-1)) + z*index];       // Index is (i-s) mod r
	t ^= t<<a;  t ^= t>>b;                                       // (I + L^a)(I + R^b)
	v ^= v<<c;  v ^= v>>d;                                       // (I + L^c)(I + R^d)
	device_x_brent[i_brent + z*index] = (v ^= t); 				 // Update circular array                 
	w += weyl;                                                   // Update Weyl generator
	
	device_x_brent[z*index + z_w] = w;
	device_x_brent[z*index + z_i_brent] = i_brent;
	
	return (v + (w^(w>>ws)));  // Return combination
	
#undef UINT64
#undef UINT32
#undef wlen
#undef r
#undef s
#undef a
#undef b
#undef c
#undef d
#undef ws 
}	

// Brent PRNG real version
__device__ double Brent_real(int index, unsigned long *device_x_brent, unsigned long seed)

{
	
#define UINT64 (sizeof(unsigned long)>>3)
#define UINT32 (1 - UINT64) 
#define UREAL64 (sizeof(double)>>3)
#define UREAL32 (1 - UREAL64)
	
	// sr = number of bits discarded = 11 for double, 40 or 8 for float
	
#define sr (11*UREAL64 +(40*UINT64 + 8*UINT32)*UREAL32)
	
	// ss (used for scaling) is 53 or 21 for double, 24 for float
	
#define ss ((53*UINT64 + 21*UINT32)*UREAL64 + 24*UREAL32)
	
	// SCALE is 0.5**ss, SC32 is 0.5**32
	
#define SCALE ((double)1/(double)((unsigned long)1<<ss)) 
#define SC32  ((double)1/((double)65536*(double)65536)) 
	
	double res;
	
	res = (double)0; 
	while (res == (double)0)  // Loop until nonzero result.
    {   // Usually only one iteration.
		res = (double)(brent_int(index, device_x_brent, seed)>>sr);     // Discard sr random bits.
		seed = (unsigned long)0;                                        // Zero seed for next time.
		if (UINT32 && UREAL64)                                          // Need another call to xor4096i.
			res += SC32*(double)brent_int(index, device_x_brent, seed); // Add low-order 32 bits.
    }
	return (SCALE*res); // Return result in (0.0, 1.0).
	
#undef UINT64
#undef UINT32
#undef UREAL64
#undef UREAL32
#undef SCALE
#undef SC32
#undef sr
#undef ss
}

// Init Brent seed
__global__ void kernel_brent_init(StackGamma stackgamma) {
	unsigned int id = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;

	if (id < stackgamma.size) {
		unsigned int seed = stackgamma.seed[id];
		float dummy = brent_int(id, stackgamma.table_x_brent, seed);
	}
}

/***********************************************************
 * Physics
 ***********************************************************/

// Compton Cross Section Per Atom (Standard - Klein-Nishina)
__device__ float Compton_CSPA_Standard(float E, unsigned short int Z) {
	float CrossSection = 0.0;
	if (Z<1 || E < 1e-4f) {return CrossSection;}

	float p1Z = Z*(2.7965e-23f + 1.9756e-27f*Z + -3.9178e-29f*Z*Z);
	float p2Z = Z*(-1.8300e-23f + -1.0205e-24f*Z + 6.8241e-27f*Z*Z);
	float p3Z = Z*(6.7527e-22f + -7.3913e-24f*Z + 6.0480e-27f*Z*Z);
	float p4Z = Z*(-1.9798e-21f + 2.7079e-24f*Z + 3.0274e-26f*Z*Z);
	float T0 = (Z < 1.5f)? 40.0e-3f : 15.0e-3f;
	float d1, d2, d3, d4, d5;

	d1 = __fdividef(fmaxf(E, T0), 0.510998910f); // X
	CrossSection = __fdividef(p1Z*__logf(1.0f+2.0f*d1), d1) + __fdividef(p2Z + p3Z*d1 + p4Z*d1*d1, 1.0f + 20.0f*d1 + 230.0f*d1*d1 + 440.0f*d1*d1*d1);

	if (E < T0) {
		d1 = __fdividef(T0+1.0e-3f, 0.510998910f); // X
		d2 = __fdividef(p1Z*__logf(1.0f+2.0f*d1), d1) + __fdividef(p2Z + p3Z*d1 + p4Z*d1*d1, 1.0f + 20.0f*d1 + 230.0f*d1*d1 + 440.0f*d1*d1*d1); // sigma
		d3 = __fdividef(-T0 * (d2 - CrossSection), CrossSection*1.0e-3f); // c1
		d4 = (Z > 1.5f)? 0.375f-0.0556f*__logf(Z) : 0.15f; // c2
		d5 = __logf(__fdividef(E, T0)); // y
		CrossSection *= __expf(-d5 * (d3 + d4*d5));
	}

	return CrossSection;
}

// PhotoElectric Cross Section Per Atom (Standard)
__device__ float PhotoElec_CSPA_Standard(float E, unsigned short int Z) {
	 // from Sandia, the same for all Z
	float Emin = fmax(PhotoElec_std_IonizationPotentials[Z]*1e-6f, 0.01e-3f);
	if (E < Emin) {return 0.0f;}
	
	int start = PhotoElec_std_CumulIntervals[Z-1];
	int stop = start + PhotoElec_std_NbIntervals[Z];
	int pos=stop;
	while (E < PhotoElec_std_SandiaTable[pos][0]*1.0e-3f){--pos;}
	float AoverAvo = 0.0103642688246f * __fdividef((float)Z, PhotoElec_std_ZtoAratio[Z]);
	float rE = __fdividef(1.0f, E);
	float rE2 = rE*rE;

	return rE * PhotoElec_std_SandiaTable[pos][1] * AoverAvo * 0.160217648e-22f
		+ rE2 * PhotoElec_std_SandiaTable[pos][2] * AoverAvo * 0.160217648e-25f
		+ rE * rE2 * PhotoElec_std_SandiaTable[pos][3] * AoverAvo * 0.160217648e-28f
		+ rE2 * rE2 * PhotoElec_std_SandiaTable[pos][4] * AoverAvo * 0.160217648e-31f;
}

// Compton Scatter (Standard, Klein-Nishina)
__device__ float3 Compton_scatter_Standard(StackGamma stack, unsigned int id) {
	float E = stack.E[id];
	float E0 = __fdividef(E, 0.510998910f);

	float epszero = __fdividef(1.0f, (1.0f + 2.0f * E0));
	float eps02 = epszero*epszero;
	float a1 = -__logf(epszero);
	float a2 = __fdividef(a1, (a1 + 0.5f*(1.0f-eps02)));

	float greject, onecost, eps, eps2, sint2, cosTheta, sinTheta, phi;
	do {
		if (a2 > Brent_real(id, stack.table_x_brent, 0)) {
			eps = __expf(-a1 * Brent_real(id, stack.table_x_brent, 0));
			eps2 = eps*eps;
		} else {
			eps2 = eps02 + (1.0f - eps02) * Brent_real(id, stack.table_x_brent, 0);
			eps = sqrt(eps2);
		}
		onecost = __fdividef(1.0f - eps, eps * E0);
		sint2 = onecost * (2.0f - onecost);
		greject = 1.0f - eps * __fdividef(sint2, 1.0f + eps2);
	} while (greject < Brent_real(id, stack.table_x_brent, 0));

	E *= eps;
	stack.E[id] = E;
	
	if (E <= 1.0e-6f) { // 1 eV
		stack.live[id] = 0;
		stack.endsimu[id] = 1; // stop this particle
		return make_float3(0.0f, 0.0f, 1.0f);
	}

	cosTheta = 1.0f - onecost;
	sinTheta = sqrt(sint2);
	phi = Brent_real(id, stack.table_x_brent, 0) * twopi;

	return make_float3(sinTheta*__cosf(phi), sinTheta*__sinf(phi), cosTheta);
}

// Compute the total Compton cross section for a given material
__device__ float Compton_CS_Standard(int mat, float E) {
	float CS = 0.0f;
	int i;
	int index = mat_index[mat];
	// Model standard
	for (i = 0; i < mat_nb_elements[mat]; ++i) {
		CS += (mat_atom_num_dens[index+i] * Compton_CSPA_Standard(E, mat_mixture[index+i]));
	}
	return CS;
}

// Compute the total Compton cross section for a given material
__device__ float PhotoElec_CS_Standard(int mat, float E) {
	float CS = 0.0f;
	int i;
	int index = mat_index[mat];
	// Model standard
	for (i = 0; i < mat_nb_elements[mat]; ++i) {
		CS += (mat_atom_num_dens[index+i] * PhotoElec_CSPA_Standard(E, mat_mixture[index+i]));
	}
	return CS;
}

/***********************************************************
 * Sources
 ***********************************************************/

// Voxelized back2back source (use the Relative Activity Integral Method)
__global__ void kernel_voxelized_source_b2b(StackGamma stackgamma1, StackGamma stackgamma2,
											int3 dim_vol, float E, float size_voxel) {
	unsigned int id = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;

	if (id < stackgamma1.size) {
		float jump = (float)(dim_vol.y * dim_vol.x);
		float ind, x, y, z;
		
		float rnd = Brent_real(id, stackgamma1.table_x_brent, 0);
		//int pos = (int)(rnd * (float)nb_act);
		int pos = 0;
		while (tex1Dfetch(tex_act_val, pos) < rnd) {++pos;};
		
		// get the voxel position (x, y, z)
		ind = (float)(tex1Dfetch(tex_act_ind, pos));
		z = floor(ind / jump);
		ind -= (z * jump);
		y = floor(ind / (float)(dim_vol.x));
		x = ind - y*dim_vol.x;

		// random position inside the voxel
		x += Brent_real(id, stackgamma1.table_x_brent, 0);
		y += Brent_real(id, stackgamma1.table_x_brent, 0);
		z += Brent_real(id, stackgamma1.table_x_brent, 0);

		// must be in mm
		x *= size_voxel;
		y *= size_voxel;
		z *= size_voxel;

		// random orientation
		float phi   = Brent_real(id, stackgamma1.table_x_brent, 0);
		float theta = Brent_real(id, stackgamma1.table_x_brent, 0);
		phi   = twopi * phi;
		theta = acosf(1.0f - 2.0f*theta);
		
		// convert to cartesian
		float dx = __cosf(phi)*__sinf(theta);
		float dy = __sinf(phi)*__sinf(theta);
		float dz = __cosf(theta);

		// first gamma
		stackgamma1.dx[id] = dx;
		stackgamma1.dy[id] = dy;
		stackgamma1.dz[id] = dz;
		stackgamma1.E[id] = E;
		stackgamma1.px[id] = x;
		stackgamma1.py[id] = y;
		stackgamma1.pz[id] = z;
		stackgamma1.t[id] = 0.0f;
		stackgamma1.live[id] = 1;
		stackgamma1.endsimu[id] = 0;
		stackgamma1.interaction[id] = 0;
		stackgamma1.ct_cpt[id] = 0;
		stackgamma1.ct_pe[id] = 0;
		stackgamma1.ct_ray[id] = 0;
		// second gamma
		stackgamma2.dx[id] = -dx;
		stackgamma2.dy[id] = -dy;
		stackgamma2.dz[id] = -dz;
		stackgamma2.E[id] = E;
		stackgamma2.px[id] = x;
		stackgamma2.py[id] = y;
		stackgamma2.pz[id] = z;
		stackgamma2.t[id] = 0.0f;
		stackgamma2.live[id] = 1;
		stackgamma2.endsimu[id] = 0;
		stackgamma2.interaction[id] = 0;
		stackgamma2.ct_cpt[id] = 0;
		stackgamma2.ct_pe[id] = 0;
		stackgamma2.ct_ray[id] = 0;
	}
}

/***********************************************************
 * Tracking kernel
 ***********************************************************/

// Fictitious tracking (or delta-tracking)
__global__ void kernel_woodcock_Standard(int3 dimvol, StackGamma stackgamma, float dimvox, int most_att_mat) {
	unsigned int id = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	int jump = dimvol.x * dimvol.y;
	float3 p, p0, delta, dimvolmm, dp;
	float3 cur_CS,prob_CS;
	int3 vox;
	float path, rec_mu_maj, E, sum_CS, t;
	int i=0;
	unsigned short int mat;
	dimvolmm.x = dimvol.x * dimvox;
	dimvolmm.y = dimvol.y * dimvox;
	dimvolmm.z = dimvol.z * dimvox;
	dimvox = __fdividef(1.0f, dimvox);

	if (id < stackgamma.size && !stackgamma.endsimu[id]) {
		p0.x = stackgamma.px[id];
		p0.y = stackgamma.py[id];
		p0.z = stackgamma.pz[id];
		p.x = p0.x;
		p.y = p0.y;
		p.z = p0.z;
		delta.x = stackgamma.dx[id];
		delta.y = stackgamma.dy[id];
		delta.z = stackgamma.dz[id];
		E = stackgamma.E[id];
		t = stackgamma.t[id];
		

		// Most attenuate material
		cur_CS.x = PhotoElec_CS_Standard(most_att_mat, E);
		cur_CS.y = Compton_CS_Standard(most_att_mat, E);
		rec_mu_maj = __fdividef(1.0f, cur_CS.x + cur_CS.y);

		// init mem share
		__shared__ float CS[256][15];
		while (i<15) {CS[threadIdx.x][i] = 0.0f; ++i;}
			
		while (1) {
			// get mean path from the most attenuate material (RibBone)
			path = -__logf(Brent_real(id, stackgamma.table_x_brent, 0)) * rec_mu_maj; // mm
			
			// fly along the path
			p.x = p.x + delta.x * path;
			p.y = p.y + delta.y * path;
			p.z = p.z + delta.z * path;

			// still inside the phantom? if not
			if (p.x < 0 || p.y < 0 || p.z < 0
				|| p.x >= dimvolmm.x || p.y >= dimvolmm.y || p.z >= dimvolmm.z) {
				stackgamma.endsimu[id] = 1; // stop simulation for this one
				stackgamma.interaction[id] = 0;
				
				float dimvoxbis = __fdividef(1.0f, dimvox);
				float r=dimvoxbis*0.1;
				
				while(p.x < -r || p.y < -r || p.z < -r
				|| p.x >= dimvolmm.x+r || p.y >= dimvolmm.y+r || p.z >= dimvolmm.z+r){
					p.x = p.x - delta.x * r;
					p.y = p.y - delta.y * r;
					p.z = p.z - delta.z * r;
				}
				
				dp.x = p0.x - p.x;
				dp.y = p0.y - p.y;
				dp.z = p0.z - p.z;
				
				t += (3.33564095198e-03f * sqrt(dp.x*dp.x + dp.y*dp.y + dp.z*dp.z));
				
				stackgamma.px[id] = p.x;
				stackgamma.py[id] = p.y;
				stackgamma.pz[id] = p.z;
				stackgamma.t[id] = t;

				return;
			}
		
			// which voxel?
			vox.x = floor(p.x * dimvox);
			vox.y = floor(p.y * dimvox);
			vox.z = floor(p.z * dimvox);
			
			// get mat
			mat = tex1Dfetch(tex_phantom, vox.z*jump + vox.y*dimvol.x + vox.x);

			// Bib of sum_CS
			if (CS[threadIdx.x][mat] == 0.0f) {
				// get CS
				cur_CS.x = PhotoElec_CS_Standard(mat, E);
				cur_CS.y = Compton_CS_Standard(mat, E);
				sum_CS = cur_CS.x + cur_CS.y;
				CS[threadIdx.x][mat] = sum_CS;
			} else {
				sum_CS = CS[threadIdx.x][mat];
			}

			// Does the interaction is real?
			if (sum_CS * rec_mu_maj > Brent_real(id, stackgamma.table_x_brent, 0)) {break;}
		}

		dp.x = p0.x - p.x;
		dp.y = p0.y - p.y;
		dp.z = p0.z - p.z;
	
		t += (3.33564095198e-03f * sqrt(dp.x*dp.x + dp.y*dp.y + dp.z*dp.z));
		
		stackgamma.px[id] = p.x;
		stackgamma.py[id] = p.y;
		stackgamma.pz[id] = p.z;
		stackgamma.t[id] = t;
		
		// Select an interaction
		// Re-use CS variables to select an interaction
		prob_CS.x = __fdividef(cur_CS.x, sum_CS);                      // pe
		prob_CS.y = __fdividef(cur_CS.y, sum_CS) + prob_CS.x;  			// cpt				
		// re-use p.x as rnd variable
		p.x = Brent_real(id, stackgamma.table_x_brent, 0);
		// selecting interaction				
		if (p.x>=0 && p.x<prob_CS.x) stackgamma.interaction[id] = 1;
		if (p.x>=prob_CS.x && p.x<prob_CS.y) stackgamma.interaction[id] = 2;
	}
}

/***********************************************************
 * Interactions
 ***********************************************************/

// Kernel interactions
__global__ void kernel_interactions(StackGamma stackgamma, int3 dimvol, float dimvox) {
	unsigned int id = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	float3 dir;

	if (id < stackgamma.size && !stackgamma.endsimu[id]) {
		switch (stackgamma.interaction[id]) {
		case 0:
			// do nothing and release the thread (maybe the block if interactions are sorted)
			return;
		case 1:
			// PhotoElectric effect
			stackgamma.live[id] = 0;    // kill the particle.
			stackgamma.endsimu[id] = 1; // stop the simulation
			++stackgamma.ct_pe[id];
			return;
		case 2:
			// Compton scattering
			++stackgamma.ct_cpt[id];
			// Model standard
			dir = Compton_scatter_Standard(stackgamma, id);
			break;
		}

		//*************************************
		// Apply new direction to the particle 
		//
		float3 p = make_float3(stackgamma.dx[id], stackgamma.dy[id], stackgamma.dz[id]);
		p = deflect_particle(p, dir);
		stackgamma.dx[id] = p.x;
		stackgamma.dy[id] = p.y;
		stackgamma.dz[id] = p.z;
		
	}
}

/***********************************************************
 * Utils Host
 ***********************************************************/
// For PRNG Brent
#define UINT64 (sizeof(unsigned long)>>3)
#define UINT32 (1 - UINT64)
#define r      (4*UINT64 + 8*UINT32)
// Stack device allocation
void init_device_stackgamma(StackGamma &stackgamma, int stack_size) {
	stackgamma.size = stack_size;
	unsigned int mem_stackgamma_float = stack_size * sizeof(float);
	unsigned int mem_stackgamma_uint = stack_size * sizeof(unsigned int);
	unsigned int mem_stackgamma_char = stack_size * sizeof(char);
	unsigned int mem_brent;
	if (r == 4) {mem_brent = stack_size * 6 * sizeof(unsigned long);}
	else {mem_brent = stack_size * 10 * sizeof(unsigned long);}

	hipMalloc((void**) &stackgamma.E, mem_stackgamma_float);
	hipMalloc((void**) &stackgamma.dx, mem_stackgamma_float);
	hipMalloc((void**) &stackgamma.dy, mem_stackgamma_float);
	hipMalloc((void**) &stackgamma.dz, mem_stackgamma_float);
	hipMalloc((void**) &stackgamma.px, mem_stackgamma_float);
	hipMalloc((void**) &stackgamma.py, mem_stackgamma_float);
	hipMalloc((void**) &stackgamma.pz, mem_stackgamma_float);
	hipMalloc((void**) &stackgamma.t, mem_stackgamma_float);
	hipMalloc((void**) &stackgamma.seed, mem_stackgamma_uint);
	hipMalloc((void**) &stackgamma.table_x_brent, mem_brent);
	hipMalloc((void**) &stackgamma.interaction, mem_stackgamma_char);
	hipMalloc((void**) &stackgamma.live, mem_stackgamma_char);
	hipMalloc((void**) &stackgamma.endsimu, mem_stackgamma_char);
	hipMalloc((void**) &stackgamma.ct_cpt, mem_stackgamma_char);
	hipMalloc((void**) &stackgamma.ct_pe, mem_stackgamma_char);
	hipMalloc((void**) &stackgamma.ct_ray, mem_stackgamma_char);
	// set endsimu to one in order to force a reload of each stack
	char* tmpc = (char*)malloc(stack_size * sizeof(char));
	int n=0; while (n<stack_size) {tmpc[n] = 1; ++n;};
	hipMemcpy(stackgamma.endsimu, tmpc, stack_size * sizeof(char), hipMemcpyHostToDevice);
	free(tmpc);
}
#undef UINT64
#undef UINT32
#undef r

// Stack host allocation
void init_host_stackgamma(StackGamma &phasespace, int stack_size) {
	phasespace.size = stack_size;
	unsigned int mem_phasespace_float = stack_size * sizeof(float);
	unsigned int mem_phasespace_uint = stack_size * sizeof(unsigned int);
	unsigned int mem_phasespace_char = stack_size * sizeof(char);
	phasespace.E = (float*)malloc(mem_phasespace_float);
	phasespace.dx = (float*)malloc(mem_phasespace_float);
	phasespace.dy = (float*)malloc(mem_phasespace_float);
	phasespace.dz = (float*)malloc(mem_phasespace_float);
	phasespace.px = (float*)malloc(mem_phasespace_float);
	phasespace.py = (float*)malloc(mem_phasespace_float);
	phasespace.pz = (float*)malloc(mem_phasespace_float);
	phasespace.t = (float*)malloc(mem_phasespace_float);
	phasespace.seed = (unsigned int*)malloc(mem_phasespace_uint);
	phasespace.interaction = (unsigned char*)malloc(mem_phasespace_char);
	phasespace.live = (unsigned char*)malloc(mem_phasespace_char);
	phasespace.endsimu = (unsigned char*)malloc(mem_phasespace_char);
	phasespace.ct_cpt = (unsigned char*)malloc(mem_phasespace_char);
	phasespace.ct_pe = (unsigned char*)malloc(mem_phasespace_char);
	phasespace.ct_ray = (unsigned char*)malloc(mem_phasespace_char);
	// set endsimu to one in order to force a reload of each stack
	int n=0; while (n<stack_size) {phasespace.endsimu[n] = 1; ++n;};

}

// Copy stack from device to host
void copy_device_to_host_stackgamma(StackGamma &stackgamma, StackGamma &phasespace) {
	int stack_size = stackgamma.size;
	unsigned int mem_stackgamma_float = stack_size * sizeof(float);
	unsigned int mem_stackgamma_char = stack_size * sizeof(char);
	hipMemcpy(phasespace.E, stackgamma.E, mem_stackgamma_float, hipMemcpyDeviceToHost);
	hipMemcpy(phasespace.dx, stackgamma.dx, mem_stackgamma_float, hipMemcpyDeviceToHost);
	hipMemcpy(phasespace.dy, stackgamma.dy, mem_stackgamma_float, hipMemcpyDeviceToHost);
	hipMemcpy(phasespace.dz, stackgamma.dz, mem_stackgamma_float, hipMemcpyDeviceToHost);
	hipMemcpy(phasespace.px, stackgamma.px, mem_stackgamma_float, hipMemcpyDeviceToHost);
	hipMemcpy(phasespace.py, stackgamma.py, mem_stackgamma_float, hipMemcpyDeviceToHost);
	hipMemcpy(phasespace.pz, stackgamma.pz, mem_stackgamma_float, hipMemcpyDeviceToHost);
	hipMemcpy(phasespace.t, stackgamma.t, mem_stackgamma_float, hipMemcpyDeviceToHost);
	hipMemcpy(phasespace.endsimu, stackgamma.endsimu, mem_stackgamma_char, hipMemcpyDeviceToHost);
	hipMemcpy(phasespace.live, stackgamma.live, mem_stackgamma_char, hipMemcpyDeviceToHost);
}

// free device mem
void free_device_stackgamma(StackGamma &stackgamma) {
	hipFree(stackgamma.E);
	hipFree(stackgamma.dx);
	hipFree(stackgamma.dy);
	hipFree(stackgamma.dz);
	hipFree(stackgamma.px);
	hipFree(stackgamma.py);
	hipFree(stackgamma.pz);
	hipFree(stackgamma.t);
	hipFree(stackgamma.interaction);
	hipFree(stackgamma.live);
	hipFree(stackgamma.endsimu);
	hipFree(stackgamma.seed);
	hipFree(stackgamma.ct_cpt);
	hipFree(stackgamma.ct_pe);
	hipFree(stackgamma.ct_ray);
	hipFree(stackgamma.table_x_brent);
}

// free host mem
void free_host_stackgamma(StackGamma &phasespace) {
	free(phasespace.E);
	free(phasespace.dx);
	free(phasespace.dy);
	free(phasespace.dz);
	free(phasespace.px);
	free(phasespace.py);
	free(phasespace.pz);
	free(phasespace.t);
	free(phasespace.interaction);
	free(phasespace.live);
	free(phasespace.endsimu);
	free(phasespace.seed);
	free(phasespace.ct_cpt);
	free(phasespace.ct_pe);
	free(phasespace.ct_ray);
}

// Count nb of partice already simulated
void get_nb_particles_simulated(StackGamma &stackgamma1, StackGamma &stackgamma2,
								StackGamma &phasespace1, StackGamma &phasespace2,
								int* gamma_sim) {
								 
	int stack_size = phasespace1.size;
	copy_device_to_host_stackgamma(stackgamma1, phasespace1);
	copy_device_to_host_stackgamma(stackgamma2, phasespace2);
	int	i = 0;
	int end1, end2;
	*gamma_sim = 0;
	while (i < stack_size) {
		end1 = (int)phasespace1.endsimu[i];
		end2 = (int)phasespace2.endsimu[i];

		if (end1) {++(*gamma_sim);};
		if (end2) {++(*gamma_sim);};
		++i;
	} // i
}

// Load phantom in the tex mem
void load_phantom_in_tex(const char* filename, int3 dim_phantom) {
	int nb = dim_phantom.z * dim_phantom.y * dim_phantom.x;
	unsigned int mem_phantom = nb * sizeof(unsigned short int);
	unsigned short int* phantom = (unsigned short int*)malloc(mem_phantom);
	// Read data
	FILE * pfile = fopen(filename, "rb");
	fread(phantom, sizeof(unsigned short int), nb, pfile);
	fclose(pfile);
	// Load phantom to texture
	unsigned short int* dphantom;
	hipMalloc((void**) &dphantom, mem_phantom);
	hipMemcpy(dphantom, phantom, mem_phantom, hipMemcpyHostToDevice);
	hipBindTexture(NULL, tex_phantom, dphantom, mem_phantom);
	free(phantom);
}

// Load activities in the tex mem
void load_activities_in_tex(const char* filename_act, const char* filename_ind, int nb) {
	FILE* pfile_act = fopen(filename_act, "rb");
	unsigned int mem_act_f = nb * sizeof(float);
	unsigned int mem_act_i = nb * sizeof(unsigned int);
	// load activities values in the tex mem
	float* activities = (float*)malloc(mem_act_f);
	fread(activities, sizeof(float), nb, pfile_act);
	fclose(pfile_act);
	float* dactivities;
	hipMalloc((void**) &dactivities, mem_act_f);
	hipMemcpy(dactivities, activities, mem_act_f, hipMemcpyHostToDevice);
	hipBindTexture(NULL, tex_act_val, dactivities, mem_act_f);
	free(activities);
	// load activities indices in the tex mem
	unsigned int* index = (unsigned int*)malloc(mem_act_i);
	FILE* pfile_ind = fopen(filename_ind, "rb");
	fread(index, sizeof(unsigned int), nb, pfile_ind);
	fclose(pfile_ind);
	unsigned int* dindex;
	hipMalloc((void**) &dindex, mem_act_i);
	hipMemcpy(dindex, index, mem_act_i, hipMemcpyHostToDevice);
	hipBindTexture(NULL, tex_act_ind, dindex, mem_act_i);
	free(index);
}

// Get the number of active voxel in the voxelized source
int get_nb_active_voxel(const char* filename_act) {
	FILE* pfile_act = fopen(filename_act, "rb");
	fseek(pfile_act, 0, SEEK_END);
	int nb = ftell(pfile_act);
	nb /= 4;
	fclose(pfile_act);
	return nb;
}
