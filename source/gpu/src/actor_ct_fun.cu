#include "hip/hip_runtime.h"
#include "actor_common.cu"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <float.h>

/***********************************************************
 * Photons Physics Effects
 ***********************************************************/

//// Comptons Standard //////////////////////////////////////

// Compton Scatter (Standard - Klein-Nishina)
__device__ float Compton_ct_SampleSecondaries_Standard(StackParticle photons, 
                                                       unsigned int id,
                                                       int* count_d) {
	float gamE0 = photons.E[id];

	float E0 = __fdividef(gamE0, 0.510998910f);
    float3 gamDir0 = make_float3(photons.dx[id], photons.dy[id], photons.dz[id]);

    // sample the energy rate of the scattered gamma

	float epszero = __fdividef(1.0f, (1.0f + 2.0f * E0));
	float eps02 = epszero*epszero;
	float a1 = -__logf(epszero);
	float a2 = __fdividef(a1, (a1 + 0.5f*(1.0f-eps02)));

	float greject, onecost, eps, eps2, sint2, cosTheta, sinTheta, phi;
	do {
		if (a2 > Brent_real(id, photons.table_x_brent, 0)) {
			eps = __expf(-a1 * Brent_real(id, photons.table_x_brent, 0));
			eps2 = eps*eps;
		} else {
			eps2 = eps02 + (1.0f - eps02) * Brent_real(id, photons.table_x_brent, 0);
			eps = sqrt(eps2);
		}
		onecost = __fdividef(1.0f - eps, eps * E0);
		sint2 = onecost * (2.0f - onecost);
		greject = 1.0f - eps * __fdividef(sint2, 1.0f + eps2);
	} while (greject < Brent_real(id, photons.table_x_brent, 0));

    // scattered gamma angles

    if (sint2 < 0.0f) {sint2 = 0.0f;}
	cosTheta = 1.0f - onecost;
	sinTheta = sqrt(sint2);
	phi = Brent_real(id, photons.table_x_brent, 0) * gpu_twopi;

    // update the scattered gamma

    float3 gamDir1 = make_float3(sinTheta*__cosf(phi), sinTheta*__sinf(phi), cosTheta);
    gamDir1 = rotateUz(gamDir1, gamDir0);
    photons.dx[id] = gamDir1.x;
    photons.dy[id] = gamDir1.y;
    photons.dz[id] = gamDir1.z;
    float gamE1  = gamE0 * eps;
    if (gamE1 > 1.0e-06f) {photons.E[id] = gamE1;}
    else {
        photons.endsimu[id] = 1; // stop this particle
        photons.active[id] = 0;  // this particle is absorbed
        atomicAdd(count_d, 1);   // count simulated primaries
        return gamE1;            // Local energy deposit
    }

    return 0.0f;
}

//// PhotoElectric Standard //////////////////////////////////////

// Compute secondaries particles
__device__ float PhotoElec_ct_SampleSecondaries_Standard(StackParticle photons,
                                                         unsigned int id,
                                                         int* count_d) {
    // Absorbed the photon
    photons.endsimu[id] = 1; // stop the simulation
    photons.active[id] = 0;  // this particle is absorbed
    atomicAdd(count_d, 1);   // count simulated primaries

    return 0.0f;
        
}

/***********************************************************
 * Tracking Kernel
 ***********************************************************/

// Photons - regular tracking
#define PHOTON_PHOTOELECTRIC 1
#define PHOTON_COMPTON 2
#define PHOTON_STEP_LIMITER 3
#define PHOTON_BOUNDARY_VOXEL 4
template <typename T1>
__global__ void kernel_ct_navigation_regular(StackParticle photons,
                                             Volume<T1> phantom,
                                             Materials materials,
                                             int* count_d) {
    unsigned int id = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;

    if (id >= photons.size) return;
    if (photons.endsimu[id]) return;
    if (!photons.active[id]) return;

    //// Init ///////////////////////////////////////////////////////////////////

    // Read position
    float3 position; // mm
    position.x = photons.px[id];
    position.y = photons.py[id];
    position.z = photons.pz[id];

    // Defined index phantom
    int4 index_phantom;
    float3 ivoxsize = inverse_vector(phantom.voxel_size);
    index_phantom.x = int(position.x * ivoxsize.x);
    index_phantom.y = int(position.y * ivoxsize.y);
    index_phantom.z = int(position.z * ivoxsize.z);
    index_phantom.w = index_phantom.z*phantom.nb_voxel_slice
                     + index_phantom.y*phantom.size_in_vox.x
                     + index_phantom.x; // linear index

    // Read direction
    float3 direction;
    direction.x = photons.dx[id];
    direction.y = photons.dy[id];
    direction.z = photons.dz[id];

    // Get energy
    float energy = photons.E[id];

    // Get material
    T1 mat = phantom.data[index_phantom.w];

    /*
	int index = materials.index[mat];
    printf("nb_mat %i mat %i index %i nb_elts %i\n", materials.nb_materials, mat, index, materials.nb_elements[mat]);

    int toto=0;
    while (toto<2) {
        printf("mixture: %i num_dens %e\n", materials.mixture[index+toto], materials.atom_num_dens[index+toto]);
        ++toto;
    }

    toto=0;
    while(toto<materials.nb_elements_total) {
        printf("elts %i\n", materials.mixture[toto]);
        ++toto;
    }
    */

    //// Find next discrete interaction ///////////////////////////////////////

    // Find next discrete interaction, total_dedx and next discrete intraction distance
    float next_interaction_distance =  FLT_MAX;
    unsigned char next_discrete_process = 0; 
    float interaction_distance;
    float cross_section;

    // Photoelectric
    cross_section = PhotoElec_CS_Standard(materials, mat, energy);
    interaction_distance = __fdividef(-__logf(Brent_real(id, photons.table_x_brent, 0)),
                                     cross_section);
    if (interaction_distance < next_interaction_distance) {
       next_interaction_distance = interaction_distance;
       next_discrete_process = PHOTON_PHOTOELECTRIC;
    }

    // Compton
    cross_section = Compton_CS_Standard(materials, mat, energy);
    interaction_distance = __fdividef(-__logf(Brent_real(id, photons.table_x_brent, 0)),
                                     cross_section);
    if (interaction_distance < next_interaction_distance) {
       next_interaction_distance = interaction_distance;
       next_discrete_process = PHOTON_COMPTON;
    }

    // Step limiter
    interaction_distance = 10.0f; // FIXME step limiter
    if (interaction_distance < next_interaction_distance) {
       next_interaction_distance = interaction_distance;
       next_discrete_process = PHOTON_STEP_LIMITER;
    }

    // Distance to the next voxel boundary (raycasting)
    interaction_distance = get_boundary_voxel_by_raycasting(index_phantom, position, 
                                                            direction, phantom.voxel_size);
    if (interaction_distance < next_interaction_distance) {
      next_interaction_distance = interaction_distance;
      next_discrete_process = PHOTON_BOUNDARY_VOXEL;
    }


    //// Move particle //////////////////////////////////////////////////////

    position.x += direction.x * next_interaction_distance;
    position.y += direction.y * next_interaction_distance;
    position.z += direction.z * next_interaction_distance;
    // Dirty part FIXME
    //   apply "magnetic grid" on the particle position due to aproximation 
    //   from the GPU (on the next_interaction_distance).
    float eps = 1.0e-6f; // 1 um
    float res_min, res_max, grid_pos_min, grid_pos_max;
    index_phantom.x = int(position.x * ivoxsize.x);
    index_phantom.y = int(position.y * ivoxsize.y);
    index_phantom.z = int(position.z * ivoxsize.z);
    // on x 
    grid_pos_min = index_phantom.x * phantom.voxel_size.x;
    grid_pos_max = (index_phantom.x+1) * phantom.voxel_size.x;
    res_min = position.x - grid_pos_min;
    res_max = position.x - grid_pos_max;
    if (res_min < eps) {position.x = grid_pos_min;}
    if (res_max > eps) {position.x = grid_pos_max;}
    // on y
    grid_pos_min = index_phantom.y * phantom.voxel_size.y;
    grid_pos_max = (index_phantom.y+1) * phantom.voxel_size.y;
    res_min = position.y - grid_pos_min;
    res_max = position.y - grid_pos_max;
    if (res_min < eps) {position.y = grid_pos_min;}
    if (res_max > eps) {position.y = grid_pos_max;}
    // on z
    grid_pos_min = index_phantom.z * phantom.voxel_size.z;
    grid_pos_max = (index_phantom.z+1) * phantom.voxel_size.z;
    res_min = position.z - grid_pos_min;
    res_max = position.z - grid_pos_max;
    if (res_min < eps) {position.z = grid_pos_min;}
    if (res_max > eps) {position.z = grid_pos_max;}

    photons.px[id] = position.x;
    photons.py[id] = position.y;
    photons.pz[id] = position.z;

    // Stop simulation if out of phantom or no more energy
    if ( position.x <= 0 || position.x >= phantom.size_in_mm.x
     || position.y <= 0 || position.y >= phantom.size_in_mm.y 
     || position.z <= 0 || position.z >= phantom.size_in_mm.z ) {
       photons.endsimu[id] = 1;                     // stop the simulation
       atomicAdd(count_d, 1);                       // count simulated primaries
       return;
    }

    //// Resolve discrete processe //////////////////////////////////////////

    // Resolve discrete processes
    if (next_discrete_process == PHOTON_PHOTOELECTRIC) {
       float discrete_loss = PhotoElec_ct_SampleSecondaries_Standard(photons, id, count_d);
       //printf("id %i PE\n", id);
    }

    if (next_discrete_process == PHOTON_COMPTON) {
       float discrete_loss = Compton_ct_SampleSecondaries_Standard(photons, id, count_d);
       //printf("id %i Compton\n", id);
    }
}
#undef PHOTON_PHOTOELECTRIC
#undef PHOTON_COMPTON
#undef PHOTON_STEP_LIMITER
#undef PHOTON_BOUNDARY_VOXEL


